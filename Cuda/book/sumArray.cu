#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void sumArray(uint8_t *arr, int *result) {
    int sum = 0;

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < 65) {
        sum = arr[tid];
    }

    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    if (threadIdx.x % 32 == 0) {
        atomicAdd(result, sum);
    }
}

int main(int argc, char **argv) {
    uint8_t source[] = {
        0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18,
        19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32, 33, 34, 35,
        36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47, 48, 49, 50, 51, 52, 53,
        54, 55, 56, 57, 58, 59, 60, 61, 62, 63, 64,
    };

    uint8_t *d_source;
    int *d_result;
    int h_result = 0;

    hipMalloc((void**)&d_source, sizeof(source));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_source, source, sizeof(source), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    sumArray<<<1, 65>>>(d_source, d_result);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_source);
    hipFree(d_result);

    printf("Sum: %d\n", h_result);

    return 0;
}
