
#include <hip/hip_runtime.h>
#include <cstdio>
int main() {
    hipDeviceProp_t cuda_props;

    int count;
    hipGetDeviceCount( &count );
    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties( &cuda_props, i );
        printf("Prop: %s\n", cuda_props.name);
    }

    return 0;
}
