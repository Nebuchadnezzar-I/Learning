
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

void __global__ print_by_gpu( int *source ) {
    source += 1;
}

int main() {
    int dest;
    int source;

    hipMalloc((void**)&source, sizeof(int));

    print_by_gpu<<<1, 10>>>(&source);

    hipMemcpy(
        &dest, &source,
        sizeof(int), hipMemcpyDeviceToHost
    );

    printf("Copied: %d \n", dest);

    hipFree( &source );

    return 0;
}
