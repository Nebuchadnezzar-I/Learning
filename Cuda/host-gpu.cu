
#include <hip/hip_runtime.h>
#include <cstdio>

void __global__ print_by_gpu( int *source ) {
    source += 1;
}

int main() {
    int dest;
    int source;

    hipMalloc((void**)&source, sizeof(int));

    print_by_gpu<<<1, 10>>>(&source);

    hipMemcpy(
        &dest, &source,
        sizeof(int), hipMemcpyDeviceToHost
    );

    hipFree( &source );

    return 0;
}
