
#include <hip/hip_runtime.h>
__global__ void print_by_gpu(int *i) { }

int main() {
    int* source;

    size_t size = 1000000 * sizeof(int);
    hipMalloc((void**)&source, size);
    hipMemset(source, 0, size);

    print_by_gpu<<<1, 1>>>(source);

    hipFree(source);
    hipDeviceSynchronize();

    return 0;
}
